
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
    printf("GPU: Hello world! \n");
}

int main() {
    printf("CPU: Hello world! \n");
    hello_world<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}