#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "cuda_fresh.h"

void sumArray(const float *a, const float *b, float *res, const int size) {
    for (int i = 0; i < size; i += 4) {
        res[i] = a[i] + b[i];
        res[i + 1] = a[i + 1] + b[i + 1];
        res[i + 2] = a[i + 2] + b[i + 2];
        res[i + 3] = a[i + 3] + b[i + 3];
    }
}

__global__ void sumArrayGPU(const float *a, const float *b, float *res) {
    auto blockId = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    auto threadId = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    auto M = blockDim.x * blockDim.y * blockDim.z;
    int idx = int(threadId + M * blockId);
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    printf("idx: %d, i: %d", idx, i);
    res[idx] = a[idx] + b[idx];
}

int main() {
    int dev = 0;
    hipSetDevice(dev);

    int nElem = 1 << 14;
    printf("Vector size: %d \n", nElem);
    int nByte = sizeof(float) * nElem;
    float *a_h = (float *)malloc(nByte);
    float *b_h = (float *)malloc(nByte);
    float *res_h= (float *) malloc(nByte);
    float *res_from_gpu_h = (float *)malloc(nByte);
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    float *a_d, *b_d, *res_d;
    CHECK(hipMalloc((float **)&a_d, nByte));
    CHECK(hipMalloc((float **)&b_d, nByte));
    CHECK(hipMalloc((float **)&res_d, nByte));

    initialData(a_h, nElem);
    initialData(b_h, nElem);

    sumArray(a_h,b_h,res_h,nElem);

    CHECK(hipMemcpy(a_d, a_h, nByte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d, b_h, nByte, hipMemcpyHostToDevice));

    dim3 block(1024);
    dim3 grid(nElem / block.x);
    sumArrayGPU<<<grid, block>>>(a_d, b_d, res_d);
    printf("Execution configuration<<<%d, %d>>> \n", grid.x, block.x);
    CHECK(hipMemcpy(res_from_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));

    checkResult(res_h, res_from_gpu_h,nElem);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);

    return 0;
}

