#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "cuda_fresh.h"

__global__ void printThreadIndex(const float *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    printf("thread_id:(%d, %d), block_id: (%d, %d), coordinate: (%d, %d), global index %2d ival %2f \n",
           threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main() {
    initDevice(0);
    int nx = 8, ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // Malloc
    auto *A_host = (float *)malloc(nBytes);
    initialData(A_host, nxy);
    printMatrix(A_host, nx, ny);

    // Cuda Malloc
    float *A_dev = nullptr;
    CHECK(hipMalloc((void **)&A_dev, nBytes));

    hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);
    printThreadIndex<<<grid, block>>>(A_dev, nx, ny);

    CHECK(hipDeviceSynchronize());
    hipFree(A_dev);
    free(A_host);

    hipDeviceReset();
    return 0;
}