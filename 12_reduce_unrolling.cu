#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "./cuda_fresh.h"


// cpu实现交错配对归约计算, 递归调用
int recursiveReduce(int *data, const int size) {
    if (size == 1) return data[0];

    const int stride = size / 2;
    if (size % 2 == 1) {
        for (int i = 0; i < stride; ++i) {
            data[i] += data[i + stride];
        }
        data[0] += data[size - 1];
    }else {
        for (int i = 0; i < stride; ++i) {
            data[i] += data[i + stride];
        }
    }

    return recursiveReduce(data, stride);
}

__global__ void warmup(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    // boundry check
    if (tid >= n) return;
    int *idata= g_idata + blockIdx.x * blockDim.x;
    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnroll2(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return ;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 2;
    if (idx + blockDim.x < n)
        g_idata[idx] += g_idata[idx + blockDim.x];
    __syncthreads();
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnroll4(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return ;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 4;
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x * 1];
        g_idata[idx] += g_idata[idx + blockDim.x * 2];
        g_idata[idx] += g_idata[idx + blockDim.x * 3];
    }
    __syncthreads();
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnroll8(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return ;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 8;
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x * 1];
        g_idata[idx] += g_idata[idx + blockDim.x * 2];
        g_idata[idx] += g_idata[idx + blockDim.x * 3];
        g_idata[idx] += g_idata[idx + blockDim.x * 4];
        g_idata[idx] += g_idata[idx + blockDim.x * 5];
        g_idata[idx] += g_idata[idx + blockDim.x * 6];
        g_idata[idx] += g_idata[idx + blockDim.x * 7];
    }
    __syncthreads();
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return ;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 8;
    if (idx + blockDim.x * 7 < n) {
        g_idata[idx] += g_idata[idx + blockDim.x * 1];
        g_idata[idx] += g_idata[idx + blockDim.x * 2];
        g_idata[idx] += g_idata[idx + blockDim.x * 3];
        g_idata[idx] += g_idata[idx + blockDim.x * 4];
        g_idata[idx] += g_idata[idx + blockDim.x * 5];
        g_idata[idx] += g_idata[idx + blockDim.x * 6];
        g_idata[idx] += g_idata[idx + blockDim.x * 7];
    }
    __syncthreads();
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid < 32) {
        // volatile int类型变量是控制变量结果写回到内存，而不是存在共享内存，或者缓存中
        // 因为下一步的计算马上要用到它，如果写入缓存，可能造成下一步的读取会读到错误的数据
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];

    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarp8(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 8;
    if (idx + blockDim.x * 7< n) {
        g_idata[idx] += g_idata[idx + blockDim.x * 1];
        g_idata[idx] += g_idata[idx + blockDim.x * 2];
        g_idata[idx] += g_idata[idx + blockDim.x * 3];
        g_idata[idx] += g_idata[idx + blockDim.x * 4];
        g_idata[idx] += g_idata[idx + blockDim.x * 5];
        g_idata[idx] += g_idata[idx + blockDim.x * 6];
        g_idata[idx] += g_idata[idx + blockDim.x * 7];
    }
    __syncthreads();
    if (blockDim.x >= 1024 && tid < 512)
        idata[tid]+=idata[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256)
        idata[tid]+=idata[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128)
        idata[tid]+=idata[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64)
        idata[tid]+=idata[tid + 64];
    __syncthreads();

    //write result for this block to global mem
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];

}

template <unsigned int iBlockSize>
__global__ void reduceCompleteUnroll(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    // boundary chexk
    if (tid >= n)
        return;
    // convert global data pointer
    int *idata = g_idata + blockDim.x * blockIdx.x * 8;
    if (idx + blockDim.x * 7< n) {
        g_idata[idx] += g_idata[idx + blockDim.x * 1];
        g_idata[idx] += g_idata[idx + blockDim.x * 2];
        g_idata[idx] += g_idata[idx + blockDim.x * 3];
        g_idata[idx] += g_idata[idx + blockDim.x * 4];
        g_idata[idx] += g_idata[idx + blockDim.x * 5];
        g_idata[idx] += g_idata[idx + blockDim.x * 6];
        g_idata[idx] += g_idata[idx + blockDim.x * 7];
    }
    __syncthreads();
    if (iBlockSize >= 1024 && tid < 512)
        idata[tid]+=idata[tid + 512];
    __syncthreads();
    if (iBlockSize >= 512 && tid < 256)
        idata[tid]+=idata[tid + 256];
    __syncthreads();
    if (iBlockSize >= 256 && tid < 128)
        idata[tid]+=idata[tid + 128];
    __syncthreads();
    if (iBlockSize >= 128 && tid < 64)
        idata[tid]+=idata[tid + 64];
    __syncthreads();

    //write result for this block to global mem
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];

}

int main() {
    initDevice(0);
    bool bResult = false;

    // initialization
    int size = 1 << 24;
    printf("with array size %d ", size);

    // execution configuration
    int blocksize = 1024;
    dim3 block(blocksize, 1);
    dim3 grid((size - 1) / block.x + 1, 1);
    printf("grid %d block %d \n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *idata_host = (int *) malloc(bytes);
    int *odata_host = (int *) malloc(grid.x * sizeof(int));
    int *tmp = (int *) malloc(bytes);

    // initialize the array
    initialData_int(idata_host, size);

    memcpy(tmp, idata_host, bytes);
    double iStart, iElaps;
    int gpu_sum = 0;

    // device memory
    int *idata_dev = nullptr;
    int *odata_dev = nullptr;
    CHECK(hipMalloc((void **)&idata_dev, bytes));
    CHECK(hipMalloc((void **)&odata_dev, grid.x * sizeof(int)));

    //cpu reduction
    int cpu_sum = 0;
    iStart = cpuSecond();
    // cpu_sum = recursiveReduce(tmp, size);
    for (int i = 0; i < size; i++)
        cpu_sum += tmp[i];
    printf("cpu sum:%d \n", cpu_sum);
    iElaps = cpuSecond() - iStart;
    printf("cpu reduce elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel 0: warmup
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    warmup <<<grid.x / 2, block>>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("gpu warmup elapsed %lf ms \n", iElaps);

    // kernel 1: reduceUnrolling2
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll2<<<grid.x / 2, block>>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 2; i++)
        gpu_sum += odata_host[i];
    printf("reduceUnrolling2 elapsed %lf ms gpu_sum: %d<<<%d, %d>>> \n",
           iElaps, gpu_sum, grid.x / 2, block.x);

    // kernel 1.1: reduceUnrolling4
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll4<<<grid.x / 4, block>>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 4; i++)
        gpu_sum += odata_host[i];
    printf("reduceUnrolling4 elapsed %lf ms gpu_sum: %d<<<%d, %d>>> \n",
           iElaps, gpu_sum, grid.x / 4, block.x);

    // kernel 1.2: reduceUnrolling8
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnroll8<<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += odata_host[i];
    printf("reduceUnrolling2 elapsed %lf ms gpu_sum: %d<<<%d, %d>>> \n",
           iElaps, gpu_sum, grid.x / 8, block.x);

    // kernel 2: reduceUnrollingWarp8
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceUnrollWarp8<<<grid.x / 8, block >>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += odata_host[i];
    printf("reduceUnrollingWarp8 elapsed %lf ms gpu_sum: %d<<<%d, %d>>>\n",
           iElaps, gpu_sum, grid.x / 8, block.x);

    // kernel 3: reduceCompleteUnrollWarp8
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceCompleteUnrollWarp8 <<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += odata_host[i];
    printf("reduceCompleteUnrollWarp8 elapsed %lf ms gpu_sum: %d<<<%d, %d>>>\n",
           iElaps, gpu_sum, grid.x / 8, block.x);

    // kernel 4: reduceCompleteUnroll
    CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    switch(blocksize) {
        case 1024:
            reduceCompleteUnroll<1024><<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
            break;
        case 512:
            reduceCompleteUnroll<512><<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
            break;
        case 256:
            reduceCompleteUnroll<256><<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
            break;
        case 128:
            reduceCompleteUnroll<128><<<grid.x / 8, block>>>(idata_dev, odata_dev, size);
            break;
    }
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += odata_host[i];
    printf("reduceCompleteUnroll elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x / 8, block.x);

    // free host memory
    free(idata_host);
    free(odata_host);
    CHECK(hipFree(idata_dev));
    CHECK(hipFree(odata_dev));

    //reset device
    hipDeviceReset();

    //check the results
    if (gpu_sum == cpu_sum) {
        printf("Test success!\n");
    }
    return EXIT_SUCCESS;
}

